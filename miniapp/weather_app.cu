#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <cassert>
#include <cerrno>
#include <cstdint>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>
#include <fcntl.h>
#include <fstream>
#include <iostream>
#include <numeric>
#include <string>
#include <sys/mman.h>
#include <unistd.h>
#include <vector>

#define CUDA_CHECK(cmd)                                                                                      \
  { hipError_t err = cmd; \
  if (err != hipSuccess) {                                                                                  \
    std::cout << "CUDA error at " << __LINE__ << " " << hipGetErrorString(err) << std::endl;                \
    return -1;}                                                                                               \
  }

__constant__ size_t month_day_boundary[13] = {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335, 366};

__inline__ __device__ int get_month_from_day_of_year(int day_of_year) {
  const int total_months = 12;

  int month = total_months / 2;
  int upper_month = total_months;
  int lower_month = 0;

  // binary search in array
  while (lower_month <= upper_month) {
    if (day_of_year >= month_day_boundary[month])
      lower_month = month + 1;
    else if (day_of_year < month_day_boundary[month])
      upper_month = month - 1;
    month = int((upper_month + lower_month) / 2);
  }
  return month; // this is 0 based
}

__global__ void construct_yearly_histogram(float *input_data, int start_year, int end_year,
                                           size_t input_grid_height, size_t input_grid_width,
                                           size_t aligned_month_file_map_offset, float *histogram_data) {
  // end year is included
  const size_t hours_per_day = 24; // assumed 24 hr data
  const size_t days_per_leap_year = 366;
  const size_t months_per_year = 12;

  // sum will accumulate in register for the full grid
  size_t grid_pitch = input_grid_height * input_grid_width;
  size_t day_grid_pitch = hours_per_day * grid_pitch;

  // output mapping
  // total 366 * 24 * 721 * 1440 active threads
  size_t linear_day_hr_loc_idx = (size_t)blockDim.x * blockIdx.x + threadIdx.x;

  size_t max_active_threads = (days_per_leap_year * (int64_t)day_grid_pitch);

  size_t day_of_year = linear_day_hr_loc_idx / day_grid_pitch; // this is 0-based
  size_t hour_of_day = (linear_day_hr_loc_idx - (day_of_year * day_grid_pitch)) / grid_pitch;
  size_t grid_linearized_idx =
      linear_day_hr_loc_idx - (day_of_year * day_grid_pitch) - (hour_of_day * grid_pitch);

  size_t grid_y = grid_linearized_idx / input_grid_width;
  size_t grid_x = grid_linearized_idx % input_grid_width;

  // month is required as each file is mapped at a separate offset - for page boundary alignment
  size_t month = (size_t)get_month_from_day_of_year((int)day_of_year);

  if (linear_day_hr_loc_idx < max_active_threads) {
    float accum_sum = 0.0f;

    for (int i = 0; i <= (end_year - start_year); i++) {
      int year = i + start_year;

      size_t access_index = (((size_t)i * months_per_year + month) * aligned_month_file_map_offset) +
                            ((day_of_year - month_day_boundary[month]) * day_grid_pitch) +
                            (hour_of_day * grid_pitch) + grid_y * input_grid_width + grid_x;
      // leap year adjustment for feb
      if (day_of_year == 59) {
        if ((year % 4) == 0) {
          // leap year - read away
          accum_sum += input_data[access_index];
        }
      } else {
        accum_sum += input_data[access_index];
      }
    }
    // write out
    histogram_data[linear_day_hr_loc_idx] = accum_sum;
  }
}

int main(int argc, char *argv[]) {

  // hard coded constants for ERA5
  const int hours_per_day = 24; // assumed 24 hr data
  const int days_per_leap_year = 366;
  const int max_days_per_month = 31;
  const int months_per_year = 12;
  const int input_grid_height = 721;
  const int input_grid_width = 1440;
  int start_year = std::atoi(argv[1]);
  int end_year = std::atoi(argv[2]);
  std::string file_path = std::string(argv[3]);

  const int num_years = end_year - start_year + 1;

  size_t max_file_size =
      sizeof(float) * max_days_per_month * hours_per_day * input_grid_height * input_grid_width;

  size_t TWO_MB = 2 * 1024 * 1024;
  size_t max_aligned_file_pages = (max_file_size + TWO_MB - 1) / TWO_MB;
  size_t max_aligned_file_size = max_aligned_file_pages * TWO_MB;

  std::cout << "aligned size: " << max_aligned_file_size << std::endl;

  std::vector<size_t> file_sizes;
  std::vector<int> open_fds;

  // 2 MB aligned VA range to allocate
  size_t va_alloc_size = sizeof(float) * num_years * months_per_year * max_aligned_file_size;

  void *va_alloc = mmap(nullptr, va_alloc_size, PROT_NONE, MAP_PRIVATE | MAP_ANONYMOUS, -1, 0);

  void *running_address = va_alloc;

  std::string file_path_base = file_path;

  for (int y = start_year; y <= end_year; y++) {
    for (int k = 1; k <= months_per_year; k++) {
      char filestr_buf[10];
      sprintf(filestr_buf, "%d%02d.bin", y, k);
      std::string filename = file_path_base + "e5.accumulated_tp_1h." + std::string(filestr_buf);
      std::cout << "mapping: " << filename << std::endl;

      std::ifstream fstreamInput(filename, std::ios::binary);
      fstreamInput.seekg(0, std::ios::end);
      size_t fileByteSize = fstreamInput.tellg();
      fstreamInput.close();

      int fd = open(filename.c_str(), O_RDONLY, 0);
      if (fd == -1) {
        std::cout << "Error opening input file: " << filename << std::endl;
        return -1;
      }

      char *mapped_addr = NULL;
      // probably need 2 MB pages for perf
      mapped_addr =
          (char *)mmap((void *)running_address, fileByteSize, PROT_READ, MAP_PRIVATE | MAP_FIXED, fd, 0);

      if (mapped_addr == MAP_FAILED) {
        close(fd);
        std::cout << "Error mapping input file: " << filename << std::endl;
        return -2;
      }

      assert(mapped_addr == (char *)running_address);
      running_address = (void *)((char *)running_address + max_aligned_file_size);

      file_sizes.push_back(fileByteSize);
      open_fds.push_back(fd);
    }
  }

  // launch kernel and feed in pointer and values
  size_t hist_bins = (size_t)days_per_leap_year * hours_per_day * input_grid_height * input_grid_width;
  size_t histogram_alloc_size = hist_bins * sizeof(float);
  float *histogram_data = NULL;
  CUDA_CHECK(hipMalloc((void **)&histogram_data, histogram_alloc_size));
  CUDA_CHECK(hipMemset(histogram_data, 0, histogram_alloc_size));

  hipEvent_t start_event, stop_event;
  CUDA_CHECK(hipEventCreate(&start_event));
  CUDA_CHECK(hipEventCreate(&stop_event));

  dim3 block(1024, 1, 1);
  dim3 grid(1, 1, 1);

  grid.x = (hist_bins + block.x - 1) / block.x;

  CUDA_CHECK(hipEventRecord(start_event));
  construct_yearly_histogram<<<grid, block, 0, NULL>>>(
      reinterpret_cast<float *>(va_alloc), start_year, end_year, (size_t)input_grid_height,
      (size_t)input_grid_width, max_aligned_file_size / sizeof(float), histogram_data);

  CUDA_CHECK(hipGetLastError()); // for catching errors from launch
  CUDA_CHECK(hipEventRecord(stop_event));
  CUDA_CHECK(hipEventSynchronize(stop_event));

  float time_ms = 0.0f;
  CUDA_CHECK(hipEventElapsedTime(&time_ms, start_event, stop_event));
  std::cout << "kernel time: " << time_ms << " ms" << std::endl;

  CUDA_CHECK(hipDeviceSynchronize()); // to start reading output histogram on host

  size_t month_day_boundary[13] = {0, 31, 60, 91, 121, 152, 182, 213, 244, 274, 305, 335, 366};
  FILE *fp_log = fopen("processed_log.csv", "w");
  fprintf(fp_log, "Month,Total Precipitation (m)\n");
  std::vector<std::vector<float>> hourly_sum_per_day;

  for (int m = 0; m < months_per_year; m++) {
    size_t start_index = month_day_boundary[m] * hours_per_day * input_grid_height * input_grid_width;
    float local_sum = 0.0f;
    for (int d = 0; d < (month_day_boundary[m + 1] - month_day_boundary[m]); d++) {
      std::vector<float> hour_sum(24);
      for (int h = 0; h < hours_per_day; h++) {
        float month_sum[16] = {0.0f};
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        size_t num_items = (input_grid_height / 2) * input_grid_width;
        size_t strided_hourly_idx = start_index + (d * hours_per_day * input_grid_height * input_grid_width) +
                                    (h * input_grid_height * input_grid_width);
        float *array_start = &(histogram_data[strided_hourly_idx]);
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, array_start, month_sum, num_items);

        d_temp_storage = malloc(temp_storage_bytes); // use HMM B-)
        hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, array_start, month_sum, num_items);
        hipDeviceSynchronize();

        free(d_temp_storage);
        hour_sum[h] = month_sum[0];
      }
      hourly_sum_per_day.push_back(hour_sum);
      local_sum += std::accumulate(hour_sum.begin(), hour_sum.end(), 0.0f);
    }
    std::cout << "Month: " << (m + 1) << " Total Precip: " << local_sum << " m" << std::endl;
    fprintf(fp_log, "%d,%f\n", m + 1, local_sum);
  }

  // get per hour total rainfall for each month
  fprintf(fp_log, "Hourly average per-month\n");
  fprintf(fp_log, "Month,Hour,Total Precipitation (m)\n");
  for (int m = 0; m < months_per_year; m++) {
    for (int h = 0; h < hours_per_day; h++) {
      float hour_sum = 0.0f;
      for (int d = 0; d < (month_day_boundary[m + 1] - month_day_boundary[m]); d++) {
        hour_sum += hourly_sum_per_day[month_day_boundary[m] + d][h];
      }
      std::cout << "m: " << m + 1 << " h: " << h + 1 << " hour_sum: " << hour_sum << std::endl;
      fprintf(fp_log, "%d,%d,%f\n", m + 1, h + 1, hour_sum);
    }
    std::cout << std::endl;
  }
  fclose(fp_log);

  FILE *fp_out = fopen("yearly_aggregates.bin", "wb");
  fwrite(histogram_data, sizeof(float), hist_bins, fp_out);
  fflush(0);
  fclose(fp_out);

  CUDA_CHECK(hipEventDestroy(start_event));
  CUDA_CHECK(hipEventDestroy(stop_event));

  CUDA_CHECK(hipFree(histogram_data));

  void *unmap_address = va_alloc;
  for (int k = 1; k < argc; k++) {
    int unmap_return = munmap(unmap_address, file_sizes[k - 1]); // unmap all address
    if (unmap_return != 0) {
      std::cout << "Error unmapping VA alloc range: " << strerror(errno) << std::endl;
    }
    close(open_fds[k - 1]);
    unmap_address = (void *)((char *)unmap_address + max_aligned_file_size);
  }

  int unmap_return = munmap(va_alloc, va_alloc_size); // unmap all address
  if (unmap_return != 0) { std::cout << "Error unmapping VA alloc range: " << strerror(errno) << std::endl; }

  return 0;
}
